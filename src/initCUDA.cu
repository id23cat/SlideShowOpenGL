/*
 * initCUDA.cu
 *
 *  Created on: 27.12.2012
 *      Author: id23cat
 */
#include "initCUDA.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>



// pointer and properties for line memory
Pixel *devImage;
int imWidth;
int imHeight;
int imPitch;

Pixel *loadToDevice(int width, int height, Pixel *data, int pixelsize)
{
	checkCudaErrors(hipMalloc(&devImage, width*height*pixelsize));
	checkCudaErrors(hipMemcpy(devImage, data, width*height*pixelsize, hipMemcpyHostToDevice));
	imWidth = width;
	imHeight = height;
	imPitch = 0;

	return devImage;
}

void deleteImage(Pixel *devPtr)
{
	checkCudaErrors(hipFree(devPtr));
}
